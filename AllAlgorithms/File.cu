#include "hip/hip_runtime.h"
#include "ParallelExamples.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
__global__
void addVectors(float* A, float* B, float* C, size_t size)
{
	size_t i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < size)
	{
		C[i] = A[i] + B[i];
	}

}

__global__
void multiplyMatrix(float* A, float* B, float* C, int m, int n, int o)
{
	size_t i = threadIdx.x + blockDim.x * blockIdx.x;
	size_t j = threadIdx.y + blockDim.y * blockIdx.y;
	/*size_t k = threadIdx.y + blockDim.y * blockIdx.y;*/
	for (size_t k = 0; k < n; k++)
	{
		C[i * m + j] += A[i * m + k] *B[k*n+j];
	}

}


void addVectorSIMT(float* A, float* B, float* C, size_t size)
{
	//host program
	float* d_A = nullptr, * d_B = nullptr, * d_C = nullptr;

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	addVectors << <(size + 2047) / 2048, 2048 >> > (d_A, d_B, d_C, size);

	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

}
/* 2x3 A and 3x1 B
| a00*b00 + a01*b10 + a02*b20 |
| a10*b00 + a11*b10 + a12*b20 |
*/
void matrixMultiplySIMTNaive(float* A, float* B, float* C, int m, int n, int o)
{
	float* d_A = nullptr, * d_B = nullptr, * d_C = nullptr;
	hipMalloc((void**)&d_A, m*n);
	hipMalloc((void**)&d_B, n*o);
	hipMalloc((void**)&d_C, n*n);
	dim3 grid = { 2,2,1 };
	dim3 block = {8,8,1 };
	hipMemcpy(d_A, A, m * n, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, n * o, hipMemcpyHostToDevice);
	multiplyMatrix << <grid, block >> > (d_A, d_B, d_C, m, n, o);
	hipMemcpy(C, d_C, m*o, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

}