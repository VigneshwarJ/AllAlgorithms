#include "hip/hip_runtime.h"
#include "ParallelExamples.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
__global__
void addVectors(float* A, float* B, float* C, size_t size)
{
	size_t i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < size)
	{
		C[i] = A[i] + B[i];
	}

}

__global__
void multiplyMatrix(float* A, float* B, float* C, int m, int n, int o)
{
	size_t i = threadIdx.x + blockDim.x * blockIdx.x;
	size_t j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i<m && j < o)
	{
		float val = 0;
		for (size_t k = 0; k < n; k++)
		{
			val += A[i * m + k] *B[k*n+j];
		}
		C[i * m + j] = val;

	}
	/*size_t k = threadIdx.y + blockDim.y * blockIdx.y;*/
}


void addVectorSIMT(float* A, float* B, float* C, size_t size)
{
	//host program
	float* d_A = nullptr, * d_B = nullptr, * d_C = nullptr;

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	addVectors << <(size + 2047) / 2048, 2048 >> > (d_A, d_B, d_C, size);

	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

}
/* 2x3 A and 3x1 B
| a00*b00 + a01*b10 + a02*b20 |
| a10*b00 + a11*b10 + a12*b20 |
*/
void matrixMultiplySIMTNaive(float* A, float* B, float* C, int m, int n, int o)
{
	float* d_A = nullptr, * d_B = nullptr, * d_C = nullptr;
	hipMalloc((void**)&d_A, m*n * sizeof(float));
	hipMalloc((void**)&d_B, n*o * sizeof(float));
	hipMalloc((void**)&d_C, m*o * sizeof(float));
	dim3 threadsPerBlock(m, o);
	dim3 blocksPerGrid(1, 1);
	if (m*o > 512) {
		threadsPerBlock.x = 16;
		threadsPerBlock.y = 16;
		blocksPerGrid.x = ceil(double(m) / double(threadsPerBlock.x));
		blocksPerGrid.y = ceil(double(o) / double(threadsPerBlock.y));
	}
	hipMemcpy(d_A, A, m * n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, n * o * sizeof(float), hipMemcpyHostToDevice);
	multiplyMatrix << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, m, n, o);
	hipMemcpy(C, d_C, m*o * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

}